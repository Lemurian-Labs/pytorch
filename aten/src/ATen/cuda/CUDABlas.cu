#include <ATen/cuda/CUDABlas.h>
#include <ATen/cuda/CUDABlas.cuh>
#include <cutlass/gemm/device/gemm.h>

namespace at {
namespace cuda {
namespace blas {

template <typename T, bool tA, bool tB>
void _cutlassGemm(CUTLASS_GEMM_ARGTYPES(T)) {
  cutlass::gemm::device::Gemm<
    T,
    tA ? cutlass::layout::ColumnMajor : cutlass::layout::RowMajor,
    T,
    tB ? cutlass::layout::ColumnMajor : cutlass::layout::RowMajor,
    T,
    cutlass::layout::RowMajor,
    at::opmath_type<T>,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm70
  > gemm_op;
  TORCH_CUTLASS_CHECK(gemm_op(
    {m, n, k},
    {a, lda},
    {b, ldb},
    {c, ldc},
    {c, ldc},
    {alpha, beta}
  ));
}

#define OP(T, _)                                                                   \
  void cutlassGemm(CUTLASS_GEMM_ARGTYPES(T)) {                                     \
    if (opa == HIPBLAS_OP_N && opb == HIPBLAS_OP_N)                                  \
      _cutlassGemm<T, false, false>(m, n, k, alpha, a, lda, b, ldb, beta, c, ldc); \
    else if (opa == HIPBLAS_OP_N)                                                   \
      _cutlassGemm<T, false, true>(m, n, k, alpha, a, lda, b, ldb, beta, c, ldc);  \
    else if (opb == HIPBLAS_OP_N)                                                   \
      _cutlassGemm<T, true, false>(m, n, k, alpha, a, lda, b, ldb, beta, c, ldc);  \
    else                                                                           \
      _cutlassGemm<T, true, true>(m, n, k, alpha, a, lda, b, ldb, beta, c, ldc);   \
  }
AT_FORALL_UNIVERSAL_TYPES(OP)
#undef OP

}
}
}
