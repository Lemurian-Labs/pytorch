#include <ATen/cuda/CUDABlas.h>
#include <ATen/cuda/CUDABlas.cuh>
#include <cutlass/gemm/device/gemm.h>
#include <c10/util/UniversalTypes.h>

namespace at {
namespace cuda {
namespace blas {

template <typename T, typename LayoutA, typename LayoutB>
void _cutlassGemm(CUTLASS_GEMM_ARGTYPES(T)) {
  using Gemm = cutlass::gemm::device::Gemm<
    T,
    LayoutA,
    T,
    LayoutB,
    T,
    cutlass::layout::RowMajor,
    T
  >;
  Gemm gemm_op;
  typename Gemm::Arguments args(
    // at::cuda::blas::gemm checks that m, n, and k are less than INT_MAX,
    // so this cast is valid
    {static_cast<int>(m), static_cast<int>(n), static_cast<int>(k)},
    {a, lda},
    {b, ldb},
    {c, ldc},
    {c, ldc},
    {alpha, beta}
  );
  TORCH_CUTLASS_CHECK(gemm_op(args));
}

#define OP(T, _)                                                                    \
  void cutlassGemm(CUTLASS_GEMM_ARGTYPES(T)) {                                      \
    if (opa == HIPBLAS_OP_N && opb == HIPBLAS_OP_N)                                   \
      _cutlassGemm<T, cutlass::layout::RowMajor, cutlass::layout::RowMajor>(        \
        opa, opb, m, n, k, alpha, a, lda, b, ldb, beta, c, ldc);                    \
    else if (opa == HIPBLAS_OP_N)                                                    \
      _cutlassGemm<T, cutlass::layout::RowMajor, cutlass::layout::ColumnMajor>(     \
        opa, opb, m, n, k, alpha, a, lda, b, ldb, beta, c, ldc);                    \
    else if (opb == HIPBLAS_OP_N)                                                    \
      _cutlassGemm<T, cutlass::layout::ColumnMajor, cutlass::layout::RowMajor>(     \
        opa, opb, m, n, k, alpha, a, lda, b, ldb, beta, c, ldc);                    \
    else                                                                            \
      _cutlassGemm<T, cutlass::layout::ColumnMajor, cutlass::layout::ColumnMajor>(  \
        opa, opb, m, n, k, alpha, a, lda, b, ldb, beta, c, ldc);                    \
  }
AT_FORALL_UNIVERSAL_TYPES(OP)
#undef OP

}
}
}
